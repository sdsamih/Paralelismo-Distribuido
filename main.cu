#include "hip/hip_runtime.h"
#include <iostream>
#include <opencv2/opencv.hpp>
#include <chrono>
#include <vector>
#include <hip/hip_runtime.h>

using namespace std;
using namespace cv;


void sequencial(string imagePath, int intervalos) {
    Mat imagem = imread(imagePath, IMREAD_GRAYSCALE);

    int linhas = imagem.rows;
    int colunas = imagem.cols;
    int tamanho_intervalo = 256 / intervalos;

    vector<int> histograma(intervalos, 0);

    //tempo inicial
    auto inicio = chrono::high_resolution_clock::now();

    for (int linha = 0; linha < linhas; linha++) {
        for (int coluna = 0; coluna < colunas; coluna++) {
            int luminescencia = imagem.at<uchar>(linha, coluna);//metodo pra acessar coordenada especifica do OpenCV
            int intervalo_escolhido = luminescencia / tamanho_intervalo;
            if (intervalo_escolhido >= intervalos)
                intervalo_escolhido = intervalos - 1;
            histograma[intervalo_escolhido]++;
        }
    }

    auto fim = chrono::high_resolution_clock::now();//tempo final
    chrono::duration<double, milli> delta = fim - inicio;//delta tempo

    cout << "Histograma (Sequencial)" << endl;
    for (int i = 0; i < intervalos; i++) {
        cout << i << ": " << histograma[i] << endl;
    }

    cout << "Tempo: " << delta.count() << " ms\n\n\n" << endl;
}

// kernel usado pra calcular o histograma (chamado dentro da funcao paralela posteriormente)
/*
imagem: Matriz carregada pelo OpenCV
linhas/colunas: Dimensões da matriz
histograma: Vetor que vai receber os valores calculados
intervalos: N de intervalos usados pro histograma
tamanho_intervalo: Tamanho de cada intervalo (calculado com base no n de intervalos (256/n_intervalos))
*/
__global__ void calcularHistogramaGPU(unsigned char* imagem, int linhas, int colunas, int* histograma, int intervalos, int tamanho_intervalo) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    int total_pixels = linhas * colunas;

    if (idx < total_pixels) {
        int luminescencia = imagem[idx];
        int intervalo_escolhido = luminescencia / tamanho_intervalo;
        if (intervalo_escolhido >= intervalos)
            intervalo_escolhido = intervalos - 1;

        atomicAdd(&histograma[intervalo_escolhido], 1);
    }
}


void paralelo(string imagePath, int intervalos) {

    //carregamento inicial da imagem igual do paralelo
    Mat imagem = imread(imagePath, IMREAD_GRAYSCALE);
    if (imagem.empty()) {
        cout << "Erro ao carregar a imagem!" << endl;
        return;
    }
    // tempo inicio
    auto inicio = chrono::high_resolution_clock::now();

    int linhas = imagem.rows;
    int colunas = imagem.cols;
    int tamanho_intervalo = 256 / intervalos;
    int total_pixels = linhas * colunas;


    // Vetor host (na ram)
    vector<int> histograma(intervalos, 0);

    // ponteiros para os espaços de vram utilizados
    unsigned char* d_imagem; //matriz da imagem
    int* d_histograma; //histograma


    hipMalloc(&d_imagem, total_pixels * sizeof(unsigned char)); //alocar os vetores (e colocar o endereco alocado nos ponteiros)
    hipMalloc(&d_histograma, intervalos * sizeof(int));

    
    hipMemcpy(d_imagem, imagem.data, total_pixels * sizeof(unsigned char), hipMemcpyHostToDevice); //Copiar a matriz da imagem pro espaco alocado
    hipMemset(d_histograma, 0, intervalos * sizeof(int)); //zerar o vetor do histograma 

    // Definir grid e block
    int threadsPorBloco = 256;
    int blocosPorGrid = (total_pixels + threadsPorBloco - 1) / threadsPorBloco; 

    

    // Chamar o kernel definido previamente (a fun)
    calcularHistogramaGPU<<<blocosPorGrid, threadsPorBloco>>>(d_imagem, linhas, colunas, d_histograma, intervalos, tamanho_intervalo);
    hipDeviceSynchronize();

    // tempo fim e delta
    auto fim = chrono::high_resolution_clock::now();
    chrono::duration<double, milli> delta = fim - inicio;

    // copiar o resultado (na vram) de volta para a ram
    hipMemcpy(histograma.data(), d_histograma, intervalos * sizeof(int), hipMemcpyDeviceToHost);

    // libera a vram
    hipFree(d_imagem);
    hipFree(d_histograma);

    // Exibir histograma
    cout << "Histograma (Paralelo CUDA):" << endl;
    for (int i = 0; i < intervalos; i++) {
        cout << i << ": " << histograma[i] << endl;
    }

    cout << "Tempo (GPU): " << delta.count() << " ms\n\n\n" << endl;
}

int main() {
    int opcao;
    string imagePath;
    int intervalos;

    cout << "Digite o caminho da imagem: ";
    cin >> imagePath;

    cout << "Digite o numero de intervalos: ";
    cin >> intervalos;

    cout << "\nEscolha o algoritmo:\n";
    cout << "1 - Sequencial\n";
    cout << "2 - Paralelo (CUDA)\n";
    cout << "Opcao: ";
    cin >> opcao;

    switch (opcao) {
        case 1:
            sequencial(imagePath, intervalos);
            break;
        case 2:
            paralelo(imagePath, intervalos);
            break;
    }

    return 0;
}
